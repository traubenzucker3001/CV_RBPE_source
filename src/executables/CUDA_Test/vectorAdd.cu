//vectorAdd aus CUDA Samples

#include <stdio.h>
#include <iostream>
#include <cmath>

// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>

#define GLM_FORCE_CUDA
#include <glm\glm.hpp>
#include <glm/gtc/quaternion.hpp>
#include <glm/gtx/quaternion.hpp>

#include "vectorAdd.cuh"
#include "vectorAdd_impl.cuh"

using namespace std;

void vectorAdd(int* A, glm::ivec4* B, int numEl){

	// Launch the Vector Add CUDA Kernel
	int threadsPerBlock = 64;	//256
	int blocksPerGrid = (numEl + threadsPerBlock - 1) / threadsPerBlock;
	printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
	//vectorAdd <<<blocksPerGrid, threadsPerBlock >>>(d_A, d_B, d_C, numElements);
	vectorAddC <<<blocksPerGrid, threadsPerBlock >>>(A, B, numEl);
}

void vectorAdd2(float* pMass, glm::vec3* pPos, glm::vec3* pVeloc, glm::vec3* pForce, float pRadius, float worldS, float springC, float dampC, glm::ivec3* pGridIndex, int* gridCounters, glm::ivec4* gridCells, int gridSL, int numEl){
	int threadsPerBlock = 64;	//256
	int blocksPerGrid = (numEl + threadsPerBlock - 1) / threadsPerBlock;
	printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);

	vectorAddCU <<<blocksPerGrid, threadsPerBlock >>>(pMass, pPos, pVeloc, pForce, pRadius, worldS, springC, dampC, pGridIndex, gridCounters, gridCells, gridSL, numEl);
}
