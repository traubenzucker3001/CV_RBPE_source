
#include <hip/hip_runtime.h>
//vectorAdd aus CUDA Samples
/*
#include <stdio.h>

// For the CUDA runtime routines (prefixed with "cuda_")
#include <cuda.h>
#include <cuda_runtime.h>
#include <device_launch_parameters.h>

#define GLM_FORCE_CUDA
#include <glm\glm.hpp>


__device__ __constant__ glm::vec3* d_A;
__device__ __constant__ glm::vec3* d_B;
__device__ __constant__ glm::vec3* d_C;

/**
 * CUDA Kernel Device code
 *
 * Computes the vector addition of A and B into C. The 3 vectors have the same
 * number of elements numElements.
 */
//__global__ void
//vectorAdd(const float *A, const float *B, float *C, int numElements)
/*__global__ void vectorAdd(glm::vec3 A, glm::vec3 B, glm::vec3 C, int numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i < 3){
		return;
	}
    //if (i < numElements){
    //    C[i] = A[i] + B[i];
    //}
	C.x = A.x + B.x;
	C.y = A.y + B.y;
	C.z = A.z + B.z;
}

/**
 * Host main routine
 */
/*int
main(void)
{
    // Error code to check return values for CUDA calls
    //cudaError_t err = cudaSuccess;

    // Print the vector length to be used, and compute its size
    int numElements = 3;
    //size_t size = numElements * sizeof(float);
	size_t size = numElements * sizeof(glm::vec3);
    printf("[Vector addition of %d elements]\n", numElements);

    // Allocate the host input vector A
    //float *h_A = (float *)malloc(size);
	glm::vec3 h_A = glm::vec3(0.0f, 0.0f, 0.0f);
    // Allocate the host input vector B
    //float *h_B = (float *)malloc(size);
	glm::vec3 h_B = glm::vec3(0.0, 0.0, 0.0);
    // Allocate the host output vector C
    //float *h_C = (float *)malloc(size);
	glm::vec3 h_C = glm::vec3(0.0, 0.0, 0.0);

    // Verify that allocations succeeded
    //if (h_A == NULL || h_B == NULL || h_C == NULL){
    //    fprintf(stderr, "Failed to allocate host vectors!\n");
    //   exit(EXIT_FAILURE);
    //}

    // Initialize the host input vectors
    //for (int i = 0; i < numElements; ++i){
        //h_A[i] = rand()/(float)RAND_MAX;
		h_A.x = rand() / (float)RAND_MAX;
		h_A.y = rand() / (float)RAND_MAX;
		h_A.z = rand() / (float)RAND_MAX;
        //h_B[i] = rand()/(float)RAND_MAX;
		h_B.x = rand() / (float)RAND_MAX;
		h_B.y = rand() / (float)RAND_MAX;
		h_B.z = rand() / (float)RAND_MAX;
   // }

    // Allocate the device input vector A
	//glm::vec3 *d_A = NULL;
    //cudaMalloc((void **)&d_A, size);
	//__device__ __constant__ glm::vec3 *d_A;
    // Allocate the device input vector B
	//glm::vec3 *d_B = NULL;
    //cudaMalloc((void **)&d_B, size);
	//__device__ __constant__ glm::vec3 *d_B;
    // Allocate the device output vector C
	//glm::vec3 *d_C = NULL;
    //cudaMalloc((void **)&d_C, size);
	//__device__ __constant__ glm::vec3 *d_C;

    // Copy the host input vectors A and B in host memory to the device input vectors in
    // device memory
    printf("Copy input data from the host memory to the CUDA device\n");
    //cudaMemcpy(d_A, h_A, size, cudaMemcpyHostToDevice);
    //cudaMemcpy(d_B, h_B, size, cudaMemcpyHostToDevice);
	cudaMemcpyToSymbol("d_A", &h_A, size); 
	cudaMemcpyToSymbol("d_B", &h_B, size);

    // Launch the Vector Add CUDA Kernel
    int threadsPerBlock = 256;
    int blocksPerGrid =(numElements + threadsPerBlock - 1) / threadsPerBlock;
    printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
    //vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, numElements);
	vectorAdd <<<blocksPerGrid, threadsPerBlock >>>(h_A, h_B, h_C, numElements);
	cudaGetLastError();

    // Copy the device result vector in device memory to the host result vector
    // in host memory.
    printf("Copy output data from the CUDA device to the host memory\n");
	//cudaMemcpy(h_C, d_C, size, cudaMemcpyDeviceToHost);
	cudaMemcpyFromSymbol(&h_C, d_C, size);

    // Verify that the result vector is correct
    for (int i = 0; i < numElements; ++i)
    {
        //if (fabs(h_A[i] + h_B[i] - h_C[i]) > 1e-5){
        //    fprintf(stderr, "Result verification failed at element %d!\n", i);
        //    exit(EXIT_FAILURE);
        //}
		if (fabs(h_A.x + h_B.x - h_C.x) > 1e-5){
			fprintf(stderr, "Result verification failed at element %d!\n", i);
			exit(EXIT_FAILURE);
		}
		if (fabs(h_A.y + h_B.y - h_C.y) > 1e-5){
			fprintf(stderr, "Result verification failed at element %d!\n", i);
			exit(EXIT_FAILURE);
		}
		if (fabs(h_A.z + h_B.z - h_C.z) > 1e-5){
			fprintf(stderr, "Result verification failed at element %d!\n", i);
			exit(EXIT_FAILURE);
		}
    }

    printf("Test PASSED\n");

    // Free device global memory
    //cudaFree(d_A);
	//cudaFree(d_B);
    //cudaFree(d_C);

    // Free host memory
    //free(h_A);
    //free(h_B);
    //free(h_C);
	//delete h_A;
	//delete h_B;
	//delete h_C;

    // Reset the device and exit
    // cudaDeviceReset causes the driver to clean up all state. While
    // not mandatory in normal operation, it is good practice.  It is also
    // needed to ensure correct operation when the application is being
    // profiled. Calling cudaDeviceReset causes all profile data to be
    // flushed before the application exits
    cudaDeviceReset();

    printf("Done\n");
    return 0;
}
*/
