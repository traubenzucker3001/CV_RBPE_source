#include "hip/hip_runtime.h"
//vectorAdd aus CUDA Samples

#include <stdio.h>
#include <cmath>

// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>

/**
* CUDA Kernel Device code
*
* Computes the vector addition of A and B into C. The 3 vectors have the same
* number of elements numElements.
*/
__global__ void vectorAdd(const float *A, const float *B, float *C, int numElements){
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i < numElements)
	{
		C[i] = A[i] + B[i];
	}
}

/**
* Host main routine
*/
int main(void){
	// Error code to check return values for CUDA calls
	//hipError_t err = hipSuccess;

	// Print the vector length to be used, and compute its size
	int numElements = 50000;
	size_t size = numElements * sizeof(float);
	printf("[Vector addition of %d elements]\n", numElements);

	// Allocate the host input (A,B) and output (C) vector
	float *h_A = (float *)malloc(size);
	float *h_B = (float *)malloc(size);
	float *h_C = (float *)malloc(size);

	// Verify that allocations succeeded
	if (h_A == NULL || h_B == NULL || h_C == NULL){
		fprintf(stderr, "Failed to allocate host vectors!\n");
		exit(EXIT_FAILURE);
	}

	// Initialize the host input vectors
	for (int i = 0; i < numElements; ++i){
		h_A[i] = rand() / (float)RAND_MAX;
		h_B[i] = rand() / (float)RAND_MAX;
	}

	// Allocate the device vectors
	float *d_A = NULL;
	hipMalloc((void **)&d_A, size);
	float *d_B = NULL;
	hipMalloc((void **)&d_B, size);
	float *d_C = NULL;
	hipMalloc((void **)&d_C, size);

	// Copy the host input vectors A and B in host memory to the device input vectors in
	// device memory
	printf("Copy input data from the host memory to the CUDA device\n");
	hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
	hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

	// Launch the Vector Add CUDA Kernel
	int threadsPerBlock = 256;
	int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;
	printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
	vectorAdd << <blocksPerGrid, threadsPerBlock >> >(d_A, d_B, d_C, numElements);
	//hipGetLastError();

	// Copy the device result vector in device memory to the host result vector
	// in host memory.
	printf("Copy output data from the CUDA device to the host memory\n");
	hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

	// Verify that the result vector is correct
	for (int i = 0; i < numElements; ++i){
		if (fabs(h_A[i] + h_B[i] - h_C[i]) > 1e-5){
			fprintf(stderr, "Result verification failed at element %d!\n", i);
			exit(EXIT_FAILURE);
		}
	}

	printf("Test PASSED\n");

	// Free device global memory
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);

	// Free host memory
	free(h_A);
	free(h_B);
	free(h_C);

	// Reset the device and exit
	// hipDeviceReset causes the driver to clean up all state. While
	// not mandatory in normal operation, it is good practice.  It is also
	// needed to ensure correct operation when the application is being
	// profiled. Calling hipDeviceReset causes all profile data to be
	// flushed before the application exits
	hipDeviceReset();

	printf("Done\n");
	return 0;
}

