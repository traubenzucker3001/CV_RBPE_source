#include "hip/hip_runtime.h"
//vectorAdd aus CUDA Samples

#include <stdio.h>
#include <iostream>
#include <cmath>

// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>

#define GLM_FORCE_CUDA
#include <glm\glm.hpp>

#include "vectorAdd.cuh"
#include "vectorAdd_impl.cuh"

using namespace std;
/**
* CUDA Kernel Device code
*
* Computes the vector addition of A and B into C. The 3 vectors have the same
* number of elements numElements.
*/
//__global__ void vectorAdd(const float *A, const float *B, float *C, int numElements){
/*__global__ void vectorAdd(glm::vec3* A, glm::vec3* B, glm::vec3* C, int numElements){
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i < numElements)
	{
		//C[i] = A[i] + B[i];
		C[i].x = A[i].x + B[i].x;
		C[i].y = A[i].y + B[i].y;
		C[i].z = A[i].z + B[i].z;
	}
}*/

/**
* Host main routine
*/
/*int main(void){
	// Error code to check return values for CUDA calls
	//hipError_t err = hipSuccess;

	// Print the vector length to be used, and compute its size
	int numElements = 50;	//50000
	//size_t size = numElements * sizeof(float);
	size_t size = numElements * sizeof(glm::vec3);
	printf("[Vector addition of %d elements]\n", numElements);

	// Allocate the host input (A,B) and output (C) vector
	//float *h_A = (float *)malloc(size);
	//float *h_B = (float *)malloc(size);
	//float *h_C = (float *)malloc(size);
	glm::vec3* h_A = (glm::vec3 *)malloc(size);
	glm::vec3* h_B = (glm::vec3 *)malloc(size);
	glm::vec3* h_C = (glm::vec3 *)malloc(size);

	// Verify that allocations succeeded
	if (h_A == NULL || h_B == NULL || h_C == NULL){
		fprintf(stderr, "Failed to allocate host vectors!\n");
		exit(EXIT_FAILURE);
	}

	// Initialize the host input vectors
	for (int i = 0; i < numElements; ++i){
		//h_A[i] = rand() / (float)RAND_MAX;
		h_A[i].x = rand() / (float)RAND_MAX;
		cout << "Ax: " << h_A[i].x << endl;
		h_A[i].y = rand() / (float)RAND_MAX;
		cout << "Ay: " << h_A[i].y << endl;
		h_A[i].z = rand() / (float)RAND_MAX;
		cout << "Az: " << h_A[i].z << endl;
		//h_B[i] = rand() / (float)RAND_MAX;
		h_B[i].x = rand() / (float)RAND_MAX;
		cout << "Bx: " << h_B[i].x << endl;
		h_B[i].y = rand() / (float)RAND_MAX;
		cout << "By: " << h_B[i].y << endl;
		h_B[i].z = rand() / (float)RAND_MAX;
		cout << "Bz: " << h_B[i].z << endl;
	}

	// Allocate the device vectors
	//float *d_A = NULL;
	glm::vec3 *d_A = NULL;
	hipMalloc((void **)&d_A, size);
	//float *d_B = NULL;
	glm::vec3 *d_B = NULL;
	hipMalloc((void **)&d_B, size);
	//float *d_C = NULL;
	glm::vec3 *d_C = NULL;
	hipMalloc((void **)&d_C, size);

	// Copy the host input vectors A and B in host memory to the device input vectors in
	// device memory
	printf("Copy input data from the host memory to the CUDA device\n");
	hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
	hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

	// Launch the Vector Add CUDA Kernel
	int threadsPerBlock = 64;	//256
	int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;
	printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
	//vectorAdd <<<blocksPerGrid, threadsPerBlock >>>(d_A, d_B, d_C, numElements);
	vectorAddC <<<blocksPerGrid, threadsPerBlock >>>(d_A, d_B, d_C, numElements);
	//hipGetLastError();

	// Copy the device result vector in device memory to the host result vector
	// in host memory.
	printf("Copy output data from the CUDA device to the host memory\n");
	hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

	// Verify that the result vector is correct
	for (int i = 0; i < numElements; ++i){
		//if (fabs(h_A[i] + h_B[i] - h_C[i]) > 1e-5){
		//	fprintf(stderr, "Result verification failed at element %d!\n", i);
		//	exit(EXIT_FAILURE);
		//}
		cout << "Cx: " << h_C[i].x << endl;
		cout << "Cy: " << h_C[i].y << endl;
		cout << "Cz: " << h_C[i].z << endl;
		if (fabs(h_A[i].x + h_B[i].x - h_C[i].x) > 1e-5){
			fprintf(stderr, "Result verification failed at element %d!\n", i);
			exit(EXIT_FAILURE);
		}
		if (fabs(h_A[i].y + h_B[i].y - h_C[i].y) > 1e-5){
			fprintf(stderr, "Result verification failed at element %d!\n", i);
			exit(EXIT_FAILURE);
		}
		if (fabs(h_A[i].z + h_B[i].z - h_C[i].z) > 1e-5){
			fprintf(stderr, "Result verification failed at element %d!\n", i);
			exit(EXIT_FAILURE);
		}
	}

	printf("Test PASSED\n");

	// Free device global memory
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);

	// Free host memory
	free(h_A);
	free(h_B);
	free(h_C);

	// Reset the device and exit
	// hipDeviceReset causes the driver to clean up all state. While
	// not mandatory in normal operation, it is good practice.  It is also
	// needed to ensure correct operation when the application is being
	// profiled. Calling hipDeviceReset causes all profile data to be
	// flushed before the application exits
	hipDeviceReset();

	printf("Done\n");
	return 0;
}*/

void vectorAdd(glm::vec3* A, glm::vec3* B, glm::vec3* C, int numEl){

	// Launch the Vector Add CUDA Kernel
	int threadsPerBlock = 64;	//256
	int blocksPerGrid = (numEl + threadsPerBlock - 1) / threadsPerBlock;
	printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
	//vectorAdd <<<blocksPerGrid, threadsPerBlock >>>(d_A, d_B, d_C, numElements);
	vectorAddC <<<blocksPerGrid, threadsPerBlock >>>(A, B, C, numEl);
}