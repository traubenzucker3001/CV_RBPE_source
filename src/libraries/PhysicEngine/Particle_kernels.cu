#include "hip/hip_runtime.h"

// <<<<<<<<<< includes >>>>>>>>>> //
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>

#include "World.h"

#define GLM_FORCE_CUDA
#include <glm\glm.hpp>

//Round a / b to nearest higher integer value
int nearHighVal(int a, int b){
	return (a % b != 0) ? (a / b + 1) : (a / b);
}

//collision detection	//unter kernel geschoben, funktion muss vor aufruf bekannt sein
/*void calcCollForces(float* pMass, glm::vec3* pPos, glm::vec3* pVeloc, glm::vec3* pForce, float pRadius, float worldS, float springC, float dampC, glm::vec3* pGridIndex, int* countGrid, glm::vec4* indexGrid, int gridSL){

	//blocks und threads berechn.
	int n = World::getInstance()->getAllPartNum();
	int blockSize = 64;
	int numThreads = (int)fmin(blockSize, n);
	int numBlocks = nearHighVal(n, numThreads);

	calcCollForcesC<<< numBlocks, numThreads >>>(pMass,pPos,pVeloc,pForce,pRadius,worldS,springC,dampC,pGridIndex,countGrid,indexGrid,gridSL);
}*/

__global__ void calcCollForcesC(float* pMass, glm::vec3* pPos, glm::vec3* pVeloc, glm::vec3* pForce, float pRadius, float worldS, float springC, float dampC, glm::vec3* pGridIndex, int* countGrid, glm::vec4* indexGrid, int gridSL){

	//TODO
	//unsigned int particleIndex = get_global_id(0);
	int pi = blockDim.x * blockIdx.x + threadIdx.x;

	pForce[pi].x = 0.0f;
	pForce[pi].y = 0.0f;
	pForce[pi].z = 0.0f;

	glm::vec3 gridIndex = pGridIndex[pi];

	//Pretend border cell is 1 position inwards to avoid checking outside bounds for neighbors
	//todo: zu cuda func
	//gridIndex = clamp(gridIndex, 1, gridSL - 2);
	gridIndex = glm::clamp(gridIndex, 1.0f, (float)gridSL - 2.0f);
	int xSteps = gridSL*gridSL;
	int ySteps = gridSL;

	int flatGridIndex = (int)gridIndex.x * xSteps + (int)gridIndex.y * ySteps + (int)gridIndex.z;

	//oder glm::vec4 besser??, dann auf umstellung bei z�hlweise achten!!
	glm::vec4 neighborCells[27];	//int4??
	int cellIndexJ = 0;

	flatGridIndex = flatGridIndex - xSteps;
	flatGridIndex = flatGridIndex + 2 * ySteps;
	flatGridIndex = flatGridIndex + 2; //zStride

	for (int x = 0; x<3; x++) {
		flatGridIndex = flatGridIndex - 3 * ySteps;

		for (int y = 0; y<3; y++) {
			flatGridIndex = flatGridIndex - 3;

			for (int z = 0; z<3; z++) {

				neighborCells[cellIndexJ] = indexGrid[flatGridIndex];
				cellIndexJ++;
				flatGridIndex++;
			}
			flatGridIndex = flatGridIndex + ySteps;
		}
		flatGridIndex = flatGridIndex + xSteps;
	}

	for (int j = 0; j<27; j++) {
		//oder glm::...
		glm::vec4 neighborParticles = glm::vec4(neighborCells[j].x,
												neighborCells[j].y,
												neighborCells[j].z,
												neighborCells[j].w );

		for (int k = 0; k<4; k++) {
			int otherParticle = (int)neighborParticles[k];
			if ((otherParticle != pi) && (otherParticle != (-1))) {
				glm::vec3 distance = pPos[otherParticle] - pPos[pi];

				float absDistance = sqrt(distance.x*distance.x + distance.y*distance.y + distance.z*distance.z);

				if ((absDistance + 0.000001f) < (2.0f * pRadius)) {
					pForce[pi].x -= springC*
						(pRadius + pRadius - absDistance)*(distance.x / absDistance);
					pForce[pi].y -= springC*
						(pRadius + pRadius - absDistance)*(distance.y / absDistance);
					pForce[pi].z -= springC*
						(pRadius + pRadius - absDistance)*(distance.z / absDistance);

					//glm::vec3
					float3 relativeVelocity = {
						pVeloc[otherParticle].x - pVeloc[pi].x,
						pVeloc[otherParticle].y - pVeloc[pi].y,
						pVeloc[otherParticle].z - pVeloc[pi].z };

					pForce[pi].x += dampC*relativeVelocity.x;
					pForce[pi].y += dampC*relativeVelocity.y;
					pForce[pi].z += dampC*relativeVelocity.z;

				}
			}
		}
	}

	//Boundary forces
	{
		bool collisionOccured = false;
		// Ground collision
		if (pPos[pi].y - pRadius < 0.0f) {
			collisionOccured = true;
			pForce[pi].y += springC*
				(pRadius - pPos[pi].y);
		}

		// X-axis Wall Collision
		if (pPos[pi].x - pRadius < -worldS) {
			collisionOccured = true;
			pForce[pi].x += springC*
				(-worldS - pPos[pi].x + pRadius);

		}
		else if (pPos[pi].x + pRadius > worldS) {
			collisionOccured = true;
			pForce[pi].x += springC*
				(worldS - pPos[pi].x - pRadius);
		}

		// Z-axis Wall Collision
		if (pPos[pi].z - pRadius < -worldS) {
			collisionOccured = true;
			pForce[pi].z += springC*
				(-worldS - pPos[pi].z + pRadius);

		}
		else if (pPos[pi].z + pRadius > worldS) {
			collisionOccured = true;
			pForce[pi].z += springC*
				(worldS - pPos[pi].z - pRadius);
		}

		// Damping
		if (collisionOccured) {
			pForce[pi].x -= dampC*pVeloc[pi].x;
			pForce[pi].y -= dampC*pVeloc[pi].y;
			pForce[pi].z -= dampC*pVeloc[pi].z;
		}
	}
}

//collision detection
void calcCollForces(float* pMass, glm::vec3* pPos, glm::vec3* pVeloc, glm::vec3* pForce, float pRadius, float worldS, float springC, float dampC, glm::vec3* pGridIndex, int* countGrid, glm::vec4* indexGrid, int gridSL){

	//blocks und threads berechn.
	int n = World::getInstance()->getAllPartNum();
	int blockSize = 64;
	int numThreads = (int)fmin(blockSize, n);
	int numBlocks = nearHighVal(n, numThreads);

	calcCollForcesC <<< numBlocks, numThreads >>>(pMass, pPos, pVeloc, pForce, pRadius, worldS, springC, dampC, pGridIndex, countGrid, indexGrid, gridSL);
}

//update particles		//unter kernel geschoben, funktion muss vor aufruf bekannt sein
//wie hei�t es in cpu version??
/*void updatePart(glm::vec3* rbPos, glm::vec3* rbVeloc, glm::mat3* rbRotMat, glm::vec3* rbAngVeloc, glm::vec3* pPos, glm::vec3* pVeloc, float pRadius){

	//blocks und threads berechn.
	int n = World::getInstance()->getAllPartNum();
	int blockSize = 64;
	int numThreads = fmin(blockSize, n);
	int numBlocks = nearHighVal(n, numThreads);

	updatePartC <<< numBlocks, numThreads >>>(rbPos, rbVeloc,rbRotMat, rbAngVeloc, pPos, pVeloc,pRadius);
}*/

__global__ void updatePartC(glm::vec3* rbPos, glm::vec3* rbVeloc, glm::mat3* rbRotMat, glm::vec3* rbAngVeloc, glm::vec3* pPos, glm::vec3* pVeloc, float pRadius){

	//weitere input param
	/*__global float* particleVBO,*/

	//TODO

	//unsigned int particleIndex = get_global_id(0);
	int pi = blockDim.x * blockIdx.x + threadIdx.x;
	int bi = pi / 27;
	//int mi = bi * 9;	//*9 nicht n�tig wenn glm::mat3!? also mi eig nit n�tig

	glm::vec3 originalRelativePos;
	//Calculate original relative position
	{
		int relativeIndex = pi % 27;

		int xIndex = relativeIndex / 9;
		relativeIndex -= xIndex * 9;

		int yIndex = relativeIndex / 3;
		relativeIndex -= yIndex * 3;

		int zIndex = relativeIndex;

		float space = 2.0f*pRadius;

		xIndex--;
		yIndex--;
		zIndex--;

		originalRelativePos.x = (float)xIndex * space;
		originalRelativePos.y = yIndex*space;
		originalRelativePos.z = zIndex*space;
	}

	//Update particle position
	{
		glm::mat3 tempRotMat = rbRotMat[bi];
		pPos[pi].x =
			originalRelativePos.x * tempRotMat[0].x +				//oder doch mi+1 +2 +3 +4 +5 +... ?!	//.xyz wollt er nit
			originalRelativePos.y * tempRotMat[0].y +
			originalRelativePos.z * tempRotMat[0].z;

		pPos[pi].y =
			originalRelativePos.x * tempRotMat[1].x +
			originalRelativePos.y * tempRotMat[1].y +
			originalRelativePos.z * tempRotMat[1].z;

		pPos[pi].z =
			originalRelativePos.x * tempRotMat[2].x +
			originalRelativePos.y * tempRotMat[2].y +
			originalRelativePos.z * tempRotMat[2].z;

		pPos[pi].x += rbPos[bi].x;
		pPos[pi].y += rbPos[bi].y;
		pPos[pi].z += rbPos[bi].z;
	}

	//Update particle velocity
	float scalar = sqrt(
		rbAngVeloc[bi].x*rbAngVeloc[bi].x +
		rbAngVeloc[bi].y*rbAngVeloc[bi].y +
		rbAngVeloc[bi].z*rbAngVeloc[bi].z);

	scalar *= scalar;

	pVeloc[pi].x = rbVeloc[bi].x;
	pVeloc[pi].y = rbVeloc[bi].y;
	pVeloc[pi].z = rbVeloc[bi].z;

	if (scalar > 0.0f) {
		float3 relativePosition = {
			pPos[pi].x - rbPos[bi].x,
			pPos[pi].y - rbPos[bi].y,
			pPos[pi].z - rbPos[bi].z };

		float scalar2 = (
			rbAngVeloc[bi].x*relativePosition.x +
			rbAngVeloc[bi].y*relativePosition.y +
			rbAngVeloc[bi].z*relativePosition.z
			) / scalar;

		float3 term = {
			relativePosition.x - rbAngVeloc[bi].x*scalar2,
			relativePosition.y - rbAngVeloc[bi].y*scalar2,
			relativePosition.z - rbAngVeloc[bi].z*scalar2 };

		pVeloc[pi].x += (rbAngVeloc[bi].y*term.z - rbAngVeloc[bi].z*term.y);
		pVeloc[pi].y += (rbAngVeloc[bi].z*term.x - rbAngVeloc[bi].x*term.z);
		pVeloc[pi].z += (rbAngVeloc[bi].x*term.y - rbAngVeloc[bi].y*term.x);
	}

	//Update particle VBO
	//siehe anhang
}

//update particles
//wie hei�t es in cpu version??
void updatePart(glm::vec3* rbPos, glm::vec3* rbVeloc, glm::mat3* rbRotMat, glm::vec3* rbAngVeloc, glm::vec3* pPos, glm::vec3* pVeloc, float pRadius){

	//blocks und threads berechn.
	int n = World::getInstance()->getAllPartNum();
	int blockSize = 64;
	int numThreads = (int)fmin(blockSize, n);
	int numBlocks = nearHighVal(n, numThreads);

	updatePartC <<< numBlocks, numThreads >>>(rbPos, rbVeloc, rbRotMat, rbAngVeloc, pPos, pVeloc, pRadius);
}

//----- anhang -----
//opencl
	//Update particle VBO
/*
	unsigned int particleVBOIndex = particleIndex * 3;
	particleVBO[particleVBOIndex] = particlePosition[particleIndex].x;
	particleVBO[particleVBOIndex + 1] = particlePosition[particleIndex].y;
	particleVBO[particleVBOIndex + 2] = particlePosition[particleIndex].z;
*/
//