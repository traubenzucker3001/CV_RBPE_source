#include "hip/hip_runtime.h"

// <<<<<<<<<< includes >>>>>>>>>> //
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>

#include <glm\glm.hpp>
#include <glm/gtc/quaternion.hpp>
#include <glm/gtx/quaternion.hpp>

#include "PhysicEngine\World.h"

//update momenta
void updateMom(float* rbMass, glm::vec3* rbForce, glm::vec3* rbPos, glm::vec3* rbLinMom, glm::vec3* rbAngMom, glm::vec3* pPos, glm::vec3* pForce, float duration, float termVeloc){

	//todo: blocks und threads berechn.
	int b = World::getInstance()->getAllBodyNum();
	int blockSize = 64;
	int numThreads = ;
	int numBlocks = ;

	updateMomC <<< numBlocks, numThreads >>>(rbMass,rbForce,rbPos,rbLinMom,rbAngMom,pPos,pForce,duration,termVeloc);
}

__global__ void updateMomC(float* rbMass, glm::vec3* rbForce, glm::vec3* rbPos, glm::vec3* rbLinMom, glm::vec3* rbAngMom, glm::vec3* pPos, glm::vec3* pForce, float duration, float termVeloc){

	//TODO
	//unsigned int bodyIndex = get_global_id(0);
	int bi = blockDim.x * blockIdx.x + threadIdx.x;

	//unsigned int totalNumberOfParticles = get_global_size(0) * 27;
	int tnop = (blockDim.x * blockIdx.x + threadIdx.x) * 27;

	int particleIndex = bi * 27;

	rbForce[bi].x = 0.0f;
	rbForce[bi].y = rbMass[bi] * -9.81f; //force of gravity
	rbForce[bi].z = 0.0f;

	glm::vec3 torque = glm::vec3( 0.0f, 0.0f, 0.0f );

	//Calculate body force and torque
	for (int i = 0; i<27; i++) {
		rbForce[bi] += pForce[particleIndex + i];
		glm::vec3 relativePos = pPos[particleIndex + i] - rbPos[bi];
		//torque += cross(relativePos, pForce[particleIndex + i]);
		torque += glm::cross(relativePos, pForce[particleIndex + i]);
	}

	float termMom = termVeloc * rbMass[bi];

	rbLinMom[bi].x += rbForce[bi].x * duration;
	rbLinMom[bi].y += rbForce[bi].y * duration;
	rbLinMom[bi].z += rbForce[bi].z * duration;

	//Limit momentum by terminal momentum
	//todo: zu cuda func
	//rbLinMom[bi] = clamp(rbLinMom[bi], -termMom, termMom);
	rbLinMom[bi] = glm::clamp(rbLinMom[bi], -termMom, termMom);

	rbAngMom[bi].x += torque.x * duration;
	rbAngMom[bi].y += torque.y * duration;
	rbAngMom[bi].z += torque.z * duration;
}


//perform step
void iterate(float* rbMass, glm::vec3* rbPos, glm::vec3* rbVeloc, glm::vec3* rbLinMom, glm::quat* rbRotQuat, glm::mat3* rbRotMat, glm::vec3* rbAngVeloc, glm::vec3* rbAngMom, glm::vec3* initIITDiago, glm::mat3* inverInertTens, float duration, float pRadius){

	//todo: blocks und threads berechn.
	int b = World::getInstance()->getAllBodyNum();
	int blockSize = 64;
	int numThreads = ;
	int numBlocks = ;

	iterateC <<< numBlocks, numThreads >>>(rbMass,rbPos,rbVeloc,rbLinMom,rbRotQuat,rbRotMat,rbAngVeloc,rbAngMom,initIITDiago,inverInertTens,duration,pRadius);
}

__global__ void iterateC(float* rbMass, glm::vec3* rbPos, glm::vec3* rbVeloc, glm::vec3* rbLinMom, glm::quat* rbRotQuat, glm::mat3* rbRotMat, glm::vec3* rbAngVeloc, glm::vec3* rbAngMom, glm::vec3* initIITDiago, glm::mat3* inverInertTens, float duration, float pRadius){

	//weitere input param
	/*__global float* bodyVBO,
	  int bodyVBOStride
	*/

	//TODO

	//unsigned int bodyIndex = get_global_id(0);
	int bi = blockDim.x * blockIdx.x + threadIdx.x;
	
	//unsigned int bodyVBOIndex = bodyIndex * 24 * 3;

	int mi = bi * 9;	//*9 nicht n�tig wenn glm::mat3!? also mi eig nit n�tig

	//Update inverse inertia tensor
	{
		glm::mat3 tempRotMat1 = rbRotMat[bi];
		glm::mat3 tempIIT1 = inverInertTens[bi];

		float a = tempRotMat1[0].x;
		float b = tempRotMat1[0].y;
		float c = tempRotMat1[0].z;
		float d = tempRotMat1[1].x;
		float e = tempRotMat1[1].y;
		float f = tempRotMat1[1].z;
		float g = tempRotMat1[2].x;
		float h = tempRotMat1[2].y;
		float i = tempRotMat1[2].z;

		float u = initIITDiago[bi].x;
		float v = initIITDiago[bi].y;
		float w = initIITDiago[bi].z;

		tempIIT1[0].x = u*a*a + b*b*v + c*c*w;
		tempIIT1[0].y = a*d*u + b*e*v + c*f*w;
		tempIIT1[0].z = a*g*u + b*h*v + c*i*w;
		tempIIT1[1].x = a*d*u + b*e*v + c*f*w;
		tempIIT1[1].y = u*d*d + e*e*v + f*f*w;
		tempIIT1[1].z = d*g*u + e*h*v + f*i*w;
		tempIIT1[2].x = a*g*u + b*h*v + c*i*w;
		tempIIT1[2].y = d*g*u + e*h*v + f*i*w;
		tempIIT1[2].z = u*g*g + h*h*v + i*i*w;
	}

	//Perform linear step
	{
		rbVeloc[bi].x = rbLinMom[bi].x / rbMass[bi];
		rbVeloc[bi].y = rbLinMom[bi].y / rbMass[bi];
		rbVeloc[bi].z = rbLinMom[bi].z / rbMass[bi];

		rbPos[bi].x += rbVeloc[bi].x * duration;
		rbPos[bi].y += rbVeloc[bi].y * duration;
		rbPos[bi].z += rbVeloc[bi].z * duration;
	}

	//Perform angular step
	{
		//Update angular velocity
		{
			glm::mat3 tempIIT2 = inverInertTens[bi];

			float a = tempIIT2[0].x;
			float b = tempIIT2[0].y;
			float c = tempIIT2[0].z;
			float d = tempIIT2[1].x;
			float e = tempIIT2[1].y;
			float f = tempIIT2[1].z;
			float g = tempIIT2[2].x;
			float h = tempIIT2[2].y;
			float i = tempIIT2[2].z;

			float u = rbAngMom[bi].x;
			float v = rbAngMom[bi].y;
			float w = rbAngMom[bi].z;

			rbAngVeloc[bi].x = a*u + b*v + c*w;
			rbAngVeloc[bi].y = d*u + e*v + f*w;
			rbAngVeloc[bi].z = g*u + h*v + i*w;
		}
		float angularVelocitySize = sqrt(
			rbAngVeloc[bi].x*rbAngVeloc[bi].x +
			rbAngVeloc[bi].y*rbAngVeloc[bi].y +
			rbAngVeloc[bi].z*rbAngVeloc[bi].z);

		if (angularVelocitySize > 0.0f) {
			float3 rotationAxis = {
				rbAngVeloc[bi].x / angularVelocitySize,
				rbAngVeloc[bi].y / angularVelocitySize,
				rbAngVeloc[bi].z / angularVelocitySize };

			float rotationAngle = angularVelocitySize*duration;

			float ds = cos(rotationAngle / 2.0f);
			float dvx = rotationAxis.x*sin(rotationAngle / 2.0f);
			float dvy = rotationAxis.y*sin(rotationAngle / 2.0f);
			float dvz = rotationAxis.z*sin(rotationAngle / 2.0f);

			float s = rbRotQuat[bi].x;
			float vx = rbRotQuat[bi].y;
			float vy = rbRotQuat[bi].z;
			float vz = rbRotQuat[bi].w;

			rbRotQuat[bi].x = s*ds - vx*dvx - vy*dvy - vz*dvz;
			rbRotQuat[bi].y = ds*vx + s*dvx + dvy*vz - dvz*vy;
			rbRotQuat[bi].z = ds*vy + s*dvy + dvz*vx - dvx*vz;
			rbRotQuat[bi].w = ds*vz + s*dvz + dvx*vy - dvy*vx;
		}
	}

	//Update rotation matrix
	{
		//Normalize quaternion
		{
			float mag2 = rbRotQuat[bi].x*rbRotQuat[bi].x +
				rbRotQuat[bi].y*rbRotQuat[bi].y +
				rbRotQuat[bi].z*rbRotQuat[bi].z +
				rbRotQuat[bi].w*rbRotQuat[bi].w;

			if (mag2 != 0.0f && (fabs(mag2 - 1.0f) > 0.00001f)) {
				float mag = sqrt(mag2);
				rbRotQuat[bi].x /= mag;
				rbRotQuat[bi].y /= mag;
				rbRotQuat[bi].z /= mag;
				rbRotQuat[bi].w /= mag;
			}
		}
		float w = rbRotQuat[bi].x;
		float x = rbRotQuat[bi].y;
		float y = rbRotQuat[bi].z;
		float z = rbRotQuat[bi].w;

		float xx = x * x;
		float yy = y * y;
		float zz = z * z;
		float xy = x * y;
		float xz = x * z;
		float yz = y * z;
		float wx = w * x;
		float wy = w * y;
		float wz = w * z;

		glm::mat3 tempRotMat2 = rbRotMat[bi];
		//WICHTIG!! --> wird auch richtiger wert beschrieben?!?! wert wird genommen und beschrieben aber eig wert in array wird ja nicht ver�ndert
		tempRotMat2[0].x = 1.0f - 2.0f*(yy + zz);
		tempRotMat2[0].y = 2.0f*(xy - wz);
		tempRotMat2[0].z = 2.0f*(xz + wy);
		tempRotMat2[1].x = 2.0f*(xy + wz);
		tempRotMat2[1].y = 1.0f - 2.0f*(xx + zz);
		tempRotMat2[1].z = 2.0f*(yz - wx);
		tempRotMat2[2].x = 2.0f*(xz - wy);
		tempRotMat2[2].y = 2.0f*(yz + wx);
		tempRotMat2[2].z = 1.0f - 2.0f*(xx + yy);

	}

	//Update body VBO
	//siehe anhang
}


//----- anhang -----
//opencl
	//Update body VBO
/*
	{

		float side = particleRadius * 3.0f;

		float normals[6][3] = { // Cube face normals 
				{ -1.0f, 0.0f, 0.0f }, { 0.0f, 1.0f, 0.0f }, { 1.0f, 0.0f, 0.0f },
				{ 0.0f, -1.0f, 0.0f }, { 0.0f, 0.0f, 1.0f }, { 0.0f, 0.0f, -1.0f } };
		int faceIndices[6][4] = { // Cube faces' vertex indices 
				{ 0, 1, 2, 3 }, { 3, 2, 6, 7 }, { 7, 6, 5, 4 },
				{ 4, 5, 1, 0 }, { 5, 6, 2, 1 }, { 7, 4, 0, 3 } };
		float vertices[8][3] = { // Cube vertex positions 
				{ -side, -side, side }, { -side, -side, -side }, { -side, side, -side },
				{ -side, side, side }, { side, -side, side }, { side, -side, -side },
				{ side, side, -side }, { side, side, side } };

		for (int i = 0; i<6; i++) { //for every face
			for (int j = 0; j<4; j++) { //for every vertex in the face

				float r0 = rotationMatrix[matrixIndex];
				float r1 = rotationMatrix[matrixIndex + 1];
				float r2 = rotationMatrix[matrixIndex + 2];
				float r3 = rotationMatrix[matrixIndex + 3];
				float r4 = rotationMatrix[matrixIndex + 4];
				float r5 = rotationMatrix[matrixIndex + 5];
				float r6 = rotationMatrix[matrixIndex + 6];
				float r7 = rotationMatrix[matrixIndex + 7];
				float r8 = rotationMatrix[matrixIndex + 8];

				float v0 = vertices[faceIndices[i][j]][0];
				float v1 = vertices[faceIndices[i][j]][1];
				float v2 = vertices[faceIndices[i][j]][2];

				bodyVBO[bodyVBOIndex] = r0*v0 + r1*v1 + r2*v2 + position[bodyIndex].x;
				bodyVBO[bodyVBOIndex + 1] = r3*v0 + r4*v1 + r5*v2 + position[bodyIndex].y;
				bodyVBO[bodyVBOIndex + 2] = r6*v0 + r7*v1 + r8*v2 + position[bodyIndex].z;

				float n0 = normals[i][0];
				float n1 = normals[i][1];
				float n2 = normals[i][2];

				bodyVBO[bodyVBOIndex + bodyVBOStride] = r0*n0 + r1*n1 + r2*n2;
				bodyVBO[bodyVBOIndex + bodyVBOStride + 1] = r3*n0 + r4*n1 + r5*n2;
				bodyVBO[bodyVBOIndex + bodyVBOStride + 2] = r6*n0 + r7*n1 + r8*n2;

				bodyVBOIndex += 3;
			}
		}
	}
}	
*/
//