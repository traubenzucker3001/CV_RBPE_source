#include "hip/hip_runtime.h"

// <<<<<<<<<< includes >>>>>>>>>> //
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>

#include <glm\glm.hpp>

//update momenta
void updateMom(){

	//blocks und threads berechn.

	updateMomC <<< numBlocks, numThreads >>>();
}

__global__ void updateMomC(){

	//TODO
}

//perform step
void iterate(){

	//blocks und threads berechn.

	iterateC <<< numBlocks, numThreads >>>();
}

__global__ void iterateC(){

	//TODO
}