#include "hip/hip_runtime.h"

// <<<<<<<<<< includes >>>>>>>>>> //
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>

#include <glm\glm.hpp>

//resetGrid
void resetGrid(int* countGrid, int4* indexGrid){

	//blocks und threads berechn.

	resetGridC<<< numBlocks, numThreads >>>(countGrid, IndexGrid);
}

__global__ void resetGridC(int* countGrid, int4* indexGrid){

	//...
	//unsigned int flatGridIndex = get_global_id(0);
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	countGrid[i] = 0;
	indexGrid[i].x = -1;
	indexGrid[i].y = -1;
	indexGrid[i].z = -1;
	indexGrid[i].w = -1;
}

//updateGRid
void updateGrid(int* contGrid, int4* indexGrid, glm::vec3* pPos, glm::vec3 gridMinPosVec, float voxelSL, int gridSL, glm::vec3* pGridIndex){

	//blocks und threads berechn.

	updateGridC <<< numBlocks, numThreads >>>(contGrid,indexGrid,pPos,gridMinPosVec,voxelSL,gridSL,pGridIndex);
}

__global__ void updateGridC(int* contGrid, int4* indexGrid, glm::vec3* pPos, glm::vec3 gridMinPosVec, float voxelSL, int gridSL, glm::vec3* pGridIndex){

	//unsigned int particleIndex = get_global_id(0);
	int pi = blockDim.x * blockIdx.x + threadIdx.x;

	pGridIndex[pi].x = (int)((pPos[pi].x - gridMinPosVec.x) / voxelSL);
	pGridIndex[pi].y = (int)((pPos[pi].y - gridMinPosVec.y) / voxelSL);
	pGridIndex[pi].z = (int)((pPos[pi].z - gridMinPosVec.z) / voxelSL);

	bool validIndex = (pGridIndex[pi].x > 0) &&
		(pGridIndex[pi].x < gridSL - 1) &&
		(pGridIndex[pi].y > 0) &&
		(pGridIndex[pi].y < gridSL - 1) &&
		(pGridIndex[pi].z > 0) &&
		(pGridIndex[pi].z < gridSL - 1);

	if (validIndex) {
		int xStride = gridSL * gridSL;
		int yStride = gridSL;
		int flatGridIndex = pGridIndex[pi].x*xStride +
			pGridIndex[pi].y * yStride +
			pGridIndex[pi].z;

		//
		int particlesInCell = atomic_inc(&countGrid[flatGridIndex]);	//?
		//

		if (particlesInCell == 3) {
			indexGrid[flatGridIndex].w = pi;
		}
		else if (particlesInCell == 2) {
			indexGrid[flatGridIndex].z = pi;
		}
		else if (particlesInCell == 1) {
			indexGrid[flatGridIndex].y = pi;
		}
		else if (particlesInCell == 0) {
			indexGrid[flatGridIndex].x = pi;
		}
	}
}