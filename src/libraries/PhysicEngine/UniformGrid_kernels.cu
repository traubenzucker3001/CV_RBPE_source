
#include <hip/hip_runtime.h>
/*
// <<<<<<<<<< includes >>>>>>>>>> //
#include <cuda.h>
#include <cuda_runtime.h>
#include <device_launch_parameters.h>

#include <glm\glm.hpp>

#include "UniformGrid.h"
#include "World.h"

/*	�nderung damit atomicInc funktioniert: 
	UniformGrid_kernel.cu --> Eigenschaften --> Konfigurationseigenschaften --> CUDA C/C++ --> Device --> Code Generation
	von "compute_20,sm_20" auf "compute_13,sm_13" ge�ndert	*/

//Round a / b to nearest higher integer value
/*int uNearHighVal(int a, int b){
	return (a % b != 0) ? (a / b + 1) : (a / b);
}	//nicht in jedem cu eine funktion, irgwo f�r alle erreichbar machen


__global__ void resetGridC(int* countGrid, glm::vec4* indexGrid){

	//cout bzw. printf() in kernel m�glich?!
	//ja, siehe --> cuda samples - 0_simple - simpleprintf

	//...
	//unsigned int flatGridIndex = get_global_id(0);
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	countGrid[i] = 0;
	indexGrid[i].x = -1;
	indexGrid[i].y = -1;
	indexGrid[i].z = -1;
	indexGrid[i].w = -1;
}

//resetGrid
void resetGrid(int* countGrid, glm::vec4* indexGrid){

	//blocks und threads berechn.
	//int b = World::getInstance()->getAllBodyNum();	//wird bodies oder particle ben�tigt, oder gitter abh�ngiges
	//thread pro gitterzelle	//wie komm ich an diese zahl?!
	/*	int g = UniformGrid::getInstance()->getGridSize();
	int blockSize = 64;
	int numThreads = (int)fmin(blockSize, g);
	int numBlocks = uNearHighVal(g, numThreads);
	//geht doch bestimmt auch noch "besser"!!?

	resetGridC <<< numBlocks, numThreads >>>(countGrid, indexGrid);
}


__global__ void updateGridC(int* countGrid, glm::vec4* indexGrid, glm::vec3* pPos, glm::vec3 gridMinPosVec, float voxelSL, int gridSL, glm::vec3* pGridIndex){

	//unsigned int particleIndex = get_global_id(0);
	int pi = blockDim.x * blockIdx.x + threadIdx.x;

	pGridIndex[pi].x = (pPos[pi].x - gridMinPosVec.x) / voxelSL;
	pGridIndex[pi].y = (pPos[pi].y - gridMinPosVec.y) / voxelSL;
	pGridIndex[pi].z = (pPos[pi].z - gridMinPosVec.z) / voxelSL;

	bool validIndex = (pGridIndex[pi].x > 0) &&
		(pGridIndex[pi].x < gridSL - 1) &&
		(pGridIndex[pi].y > 0) &&
		(pGridIndex[pi].y < gridSL - 1) &&
		(pGridIndex[pi].z > 0) &&
		(pGridIndex[pi].z < gridSL - 1);

	if (validIndex) {
		int xStride = gridSL * gridSL;
		int yStride = gridSL;
		int flatGridIndex = (int)pGridIndex[pi].x*xStride +
							(int)pGridIndex[pi].y * yStride +
							(int)pGridIndex[pi].z;

		//todo: zu cuda func
		//int particlesInCell = atomic_inc(&countGrid[flatGridIndex]);	//?
		//
		int n = 4;
		unsigned int* atom = (unsigned int*)(&countGrid[flatGridIndex]);		//potenzielle fehlerquelle
		int particlesInCell = atomicInc(atom,n);
		//

		if (particlesInCell == 3) {
			indexGrid[flatGridIndex].w = (float)pi;
		}
		else if (particlesInCell == 2) {
			indexGrid[flatGridIndex].z = (float)pi;
		}
		else if (particlesInCell == 1) {
			indexGrid[flatGridIndex].y = (float)pi;
		}
		else if (particlesInCell == 0) {
			indexGrid[flatGridIndex].x = (float)pi;
		}
	}
}

//updateGRid
void updateGrid(int* countGrid, glm::vec4* indexGrid, glm::vec3* pPos, glm::vec3 gridMinPosVec, float voxelSL, int gridSL, glm::vec3* pGridIndex){

	//blocks und threads berechn.
	//int b = World::getInstance()->getAllBodyNum();	//wird bodies oder particle ben�tigt, oder gitter abh�ngiges
	//thread pro part.
	int p = World::getInstance()->getAllPartNum();
	int blockSize = 64;
	int numThreads = (int)fmin(blockSize, p);
	int numBlocks = uNearHighVal(p, numThreads);
	//geht doch bestimmt auch noch "besser"!!?

	updateGridC <<< numBlocks, numThreads >>>(countGrid, indexGrid, pPos, gridMinPosVec, voxelSL, gridSL, pGridIndex);
}
*/