#include "hip/hip_runtime.h"

// <<<<<<<<<< includes >>>>>>>>>> //
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>

#include <glm\glm.hpp>

/*	�nderung damit atomicInc funktioniert: 
	UniformGrid_kernel.cu --> Eigenschaften --> Konfigurationseigenschaften --> CUDA C/C++ --> Device --> Code Generation
	von "compute_20,sm_20" auf "compute_13,sm_13" ge�ndert	*/

//resetGrid		//unter kernel geschoben, funktion muss vor aufruf bekannt sein
/*void resetGrid(int* countGrid, glm::vec4* indexGrid){

	//blocks und threads berechn.
	//int b = World::getInstance()->getAllBodyNum();	//wird bodies oder particle ben�tigt, oder gitter abh�ngiges
	int blockSize = 64;
	int numThreads = ;
	int numBlocks = ;

	resetGridC<<< numBlocks, numThreads >>>(countGrid, IndexGrid);
}*/

__global__ void resetGridC(int* countGrid, glm::vec4* indexGrid){

	//cout bzw. printf() in kernel m�glich?!
	//ja, siehe --> cuda samples - 0_simple - simpleprintf

	//...
	//unsigned int flatGridIndex = get_global_id(0);
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	countGrid[i] = 0;
	indexGrid[i].x = -1;
	indexGrid[i].y = -1;
	indexGrid[i].z = -1;
	indexGrid[i].w = -1;
}

//resetGrid
void resetGrid(int* countGrid, glm::vec4* indexGrid){

	//blocks und threads berechn.
	//int b = World::getInstance()->getAllBodyNum();	//wird bodies oder particle ben�tigt, oder gitter abh�ngiges
	int blockSize = 64;
	int numThreads = ;
	int numBlocks = ;

	resetGridC <<< numBlocks, numThreads >>>(countGrid, IndexGrid);
}

//updateGRid	//unter kernel geschoben, funktion muss vor aufruf bekannt sein
/*void updateGrid(int* countGrid, glm::vec4* indexGrid, glm::vec3* pPos, glm::vec3 gridMinPosVec, float voxelSL, int gridSL, glm::vec3* pGridIndex){

	//blocks und threads berechn.
	//int b = World::getInstance()->getAllBodyNum();	//wird bodies oder particle ben�tigt, oder gitter abh�ngiges
	int blockSize = 64;
	int numThreads = ;
	int numBlocks = ;

	updateGridC <<< numBlocks, numThreads >>>(countGrid,indexGrid,pPos,gridMinPosVec,voxelSL,gridSL,pGridIndex);
}*/

__global__ void updateGridC(int* countGrid, glm::vec4* indexGrid, glm::vec3* pPos, glm::vec3 gridMinPosVec, float voxelSL, int gridSL, glm::vec3* pGridIndex){

	//unsigned int particleIndex = get_global_id(0);
	int pi = blockDim.x * blockIdx.x + threadIdx.x;

	pGridIndex[pi].x = (pPos[pi].x - gridMinPosVec.x) / voxelSL;
	pGridIndex[pi].y = (pPos[pi].y - gridMinPosVec.y) / voxelSL;
	pGridIndex[pi].z = (pPos[pi].z - gridMinPosVec.z) / voxelSL;

	bool validIndex = (pGridIndex[pi].x > 0) &&
		(pGridIndex[pi].x < gridSL - 1) &&
		(pGridIndex[pi].y > 0) &&
		(pGridIndex[pi].y < gridSL - 1) &&
		(pGridIndex[pi].z > 0) &&
		(pGridIndex[pi].z < gridSL - 1);

	if (validIndex) {
		int xStride = gridSL * gridSL;
		int yStride = gridSL;
		int flatGridIndex = (int)pGridIndex[pi].x*xStride +
							(int)pGridIndex[pi].y * yStride +
							(int)pGridIndex[pi].z;

		//todo: zu cuda func
		//int particlesInCell = atomic_inc(&countGrid[flatGridIndex]);	//?
		//
		int n = 4;
		int particlesInCell = atomicInc(&countGrid[flatGridIndex],n);
		//

		if (particlesInCell == 3) {
			indexGrid[flatGridIndex].w = (float)pi;
		}
		else if (particlesInCell == 2) {
			indexGrid[flatGridIndex].z = (float)pi;
		}
		else if (particlesInCell == 1) {
			indexGrid[flatGridIndex].y = (float)pi;
		}
		else if (particlesInCell == 0) {
			indexGrid[flatGridIndex].x = (float)pi;
		}
	}
}

//updateGRid
void updateGrid(int* countGrid, glm::vec4* indexGrid, glm::vec3* pPos, glm::vec3 gridMinPosVec, float voxelSL, int gridSL, glm::vec3* pGridIndex){

	//blocks und threads berechn.
	//int b = World::getInstance()->getAllBodyNum();	//wird bodies oder particle ben�tigt, oder gitter abh�ngiges
	int blockSize = 64;
	int numThreads = ;
	int numBlocks = ;

	updateGridC <<< numBlocks, numThreads >>>(countGrid, indexGrid, pPos, gridMinPosVec, voxelSL, gridSL, pGridIndex);
}