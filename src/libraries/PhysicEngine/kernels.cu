#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define GLM_FORCE_CUDA
#include <glm\glm.hpp>

//#include "UniformGrid.h"
#include "World.h"

#include "kernels.cuh"
#include "kernels_impl.cuh"

//link fix try 4
extern World* world;


int nearHighVal(int a, int b){
	return (a % b != 0) ? (a / b + 1) : (a / b);
}

void fillDeviceSymbols(float voxelS, int gridSL, float worldS, float springC, float dampC, float pRadius, float duration, float termVeloc, glm::vec3 gridMinPosVector){
	
	hipMemcpyToSymbol((const void*)&d_voxelS, &voxelS, sizeof(float));
	
	hipMemcpyToSymbol((const void*)&d_gridSL, &gridSL, sizeof(int));
	
	hipMemcpyToSymbol((const void*)&d_worldS, &worldS, sizeof(float));
	
	hipMemcpyToSymbol((const void*)&d_springC, &springC, sizeof(float));
	
	hipMemcpyToSymbol((const void*)&d_dampC, &dampC, sizeof(float));
	
	hipMemcpyToSymbol((const void*)&d_pRadius, &pRadius, sizeof(float));
	
	hipMemcpyToSymbol((const void*)&d_duration, &duration, sizeof(float));
	
	hipMemcpyToSymbol((const void*)&d_termVeloc, &termVeloc, sizeof(float));

	//hipMemcpyToSymbol(HIP_SYMBOL("d_gridMinPosVector"), &h_gridMinPosVector, sizeof(glm::vec3));
	hipMemcpyToSymbol((const void*)&d_gridMinPosVecX, &gridMinPosVector.x, sizeof(float));
	
	hipMemcpyToSymbol((const void*)&d_gridMinPosVecY, &gridMinPosVector.y, sizeof(float));
	
	hipMemcpyToSymbol((const void*)&d_gridMinPosVecZ, &gridMinPosVector.z, sizeof(float));
}


//<<<<<<<<<< uniformgrid kernels >>>>>>>>>>	
void resetGrid(int* gridCounters, glm::ivec4* gridCells, int g){
		
	//blocks und threads berechn.
	//thread pro gitterzelle
	int blockSize = 64;	//64, 256, 512, 1024	
	//int numThreads = (int)fmin(blockSize, g);
	int numThreads = blockSize;
	int numBlocks = nearHighVal(g, numThreads);	
	//int numBlocks = ((g + 1024 -1) / 1024);
	
	//int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;	//aus vectorAdd

	resetGridC <<< numBlocks, numThreads >>>(gridCounters, gridCells, g);
	hipDeviceSynchronize();
}

void updateGrid(int* gridCounters, glm::ivec4* gridCells, glm::vec3* pPos, glm::ivec3* pGridIndex){	//, float voxelSL, int gridSL , glm::vec3 gridMinPosVec

	//thread pro part.
	int p = world->getAllPartNum();

	int blockSize = 64;
	int numThreads = (int)fmin(blockSize, p);
	int numBlocks = nearHighVal(p, numThreads);

	updateGridC <<< numBlocks, numThreads >>>(gridCounters, gridCells, pPos, pGridIndex, p);	// voxelSL, gridSL, , gridMinPosVec
	hipDeviceSynchronize();
}

//<<<<<<<<<< rigidbody kernels >>>>>>>>>>
void updateMom(float* rbMass, glm::vec3* rbForce, glm::vec3* rbPos, glm::vec3* rbLinMom, glm::vec3* rbAngMom, glm::vec3* pPos, glm::vec3* pForce){	//, float duration, float termVeloc

	//thread pro body
	int b = world->getAllBodyNum();
	int blockSize = 64;
	int numThreads = (int)fmin(blockSize, b);
	int numBlocks = nearHighVal(b, numThreads);

	updateMomC<<<numBlocks, numThreads>>>(rbMass, rbForce, rbPos, rbLinMom, rbAngMom, pPos, pForce, b);	//duration, termVeloc,
	hipDeviceSynchronize();
}

void iterate(float* rbMass, glm::vec3* rbPos, glm::vec3* rbVeloc, glm::vec3* rbLinMom, glm::quat* rbRotQuat, glm::mat3* rbRotMat, glm::vec3* rbAngVeloc, glm::vec3* rbAngMom, glm::vec3* initIITDiago, glm::mat3* inverInertTens){	//float duration, float pRadius

	//thread pro body
	int b = world->getAllBodyNum();
	int blockSize = 64;
	int numThreads = (int)fmin(blockSize, b);
	int numBlocks = nearHighVal(b, numThreads);

	iterateC<<<numBlocks, numThreads>>>(rbMass, rbPos, rbVeloc, rbLinMom, rbRotQuat, rbRotMat, rbAngVeloc, rbAngMom, initIITDiago, inverInertTens,b);	//, duration, pRadius
	hipDeviceSynchronize();
}

//<<<<<<<<<< particles kernels >>>>>>>>>>
void calcCollForces(float* pMass, glm::vec3* pPos, glm::vec3* pVeloc, glm::vec3* pForce, glm::ivec3* pGridIndex, int* gridCounters, glm::ivec4* gridCells){	//, int gridSL , float pRadius, float worldS, float springC, float dampC

	//thread pro part.
	int p = world->getAllPartNum();
		
	int blockSize = 64;
	int numThreads = (int)fmin(blockSize, p);
	int numBlocks = nearHighVal(p, numThreads);
	
	calcCollForcesC <<< numBlocks, numThreads >>>(pMass, pPos, pVeloc, pForce, pGridIndex, gridCounters, gridCells, p);	// gridSL, , pRadius, worldS, springC, dampC
	hipDeviceSynchronize();
}

void updatePart(glm::vec3* rbPos, glm::vec3* rbVeloc, glm::mat3* rbRotMat, glm::vec3* rbAngVeloc, glm::vec3* pPos, glm::vec3* pVeloc){	//, float pRadius

	//threads pro part.
	int p = world->getAllPartNum();
	int blockSize = 64;
	int numThreads = (int)fmin(blockSize, p);
	int numBlocks = nearHighVal(p, numThreads);

	updatePartC <<< numBlocks, numThreads >>>(rbPos, rbVeloc, rbRotMat, rbAngVeloc, pPos, pVeloc, p);	//pRadius,
	hipDeviceSynchronize();
}
