#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define GLM_FORCE_CUDA
#include <glm\glm.hpp>

//#include "UniformGrid.h"
#include "World.h"
#include "DemoApp\Demo.h"

#include "kernels.cuh"
#include "kernels_impl.cuh"

//#include <ctime>
//#include <chrono>
#include <windows.h>

//link fix try 4
extern World* world;
extern Demo* demo;


int nearHighVal(int a, int b){
	return (a % b != 0) ? (a / b + 1) : (a / b);
}

void fillDeviceSymbols(float voxelS, int gridSL, float worldS, float springC, float dampC, float pRadius, float duration, float termVeloc, glm::vec3 gridMinPosVector){
	
	hipMemcpyToSymbol((const void*)&d_voxelS, &voxelS, sizeof(float));
	
	hipMemcpyToSymbol((const void*)&d_gridSL, &gridSL, sizeof(int));
	
	hipMemcpyToSymbol((const void*)&d_worldS, &worldS, sizeof(float));
	
	hipMemcpyToSymbol((const void*)&d_springC, &springC, sizeof(float));
	
	hipMemcpyToSymbol((const void*)&d_dampC, &dampC, sizeof(float));
	
	hipMemcpyToSymbol((const void*)&d_pRadius, &pRadius, sizeof(float));
	
	hipMemcpyToSymbol((const void*)&d_duration, &duration, sizeof(float));
	
	hipMemcpyToSymbol((const void*)&d_termVeloc, &termVeloc, sizeof(float));

	//hipMemcpyToSymbol(HIP_SYMBOL("d_gridMinPosVector"), &h_gridMinPosVector, sizeof(glm::vec3));
	hipMemcpyToSymbol((const void*)&d_gridMinPosVecX, &gridMinPosVector.x, sizeof(float));
	
	hipMemcpyToSymbol((const void*)&d_gridMinPosVecY, &gridMinPosVector.y, sizeof(float));
	
	hipMemcpyToSymbol((const void*)&d_gridMinPosVecZ, &gridMinPosVector.z, sizeof(float));
}


//<<<<<<<<<< uniformgrid kernels >>>>>>>>>>	
void resetGrid(int* gridCounters, glm::ivec4* gridCells, int g){
		
	LARGE_INTEGER StartingTime, EndingTime, ElapsedMicroseconds;
	LARGE_INTEGER Frequency;

	//blocks und threads berechn.
	//thread pro gitterzelle
	int blockSize = 128;	//64, 128, 256, 512, 1024	
	int numThreads = blockSize;
	int numBlocks = nearHighVal(g, numThreads);	

	//int numThreads = (int)fmin(blockSize, g);
	//int numBlocks = ((g + 1024 -1) / 1024);	
	//int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;	//aus vectorAdd
	
	QueryPerformanceFrequency(&Frequency);
	QueryPerformanceCounter(&StartingTime);

	resetGridC <<< numBlocks, numThreads >>>(gridCounters, gridCells, g);
	hipDeviceSynchronize();

	QueryPerformanceCounter(&EndingTime);
	ElapsedMicroseconds.QuadPart = EndingTime.QuadPart - StartingTime.QuadPart;
	//cout << ElapsedMicroseconds.QuadPart << endl;
	ElapsedMicroseconds.QuadPart *= 1000000;
	ElapsedMicroseconds.QuadPart /= Frequency.QuadPart;
	//cout << ElapsedMicroseconds.QuadPart << endl;
	demo->file1 << ElapsedMicroseconds.QuadPart * 0.000001 << endl;
}

void updateGrid(int* gridCounters, glm::ivec4* gridCells, glm::vec3* pPos, glm::ivec3* pGridIndex){	//, float voxelSL, int gridSL , glm::vec3 gridMinPosVec

	LARGE_INTEGER StartingTime, EndingTime, ElapsedMicroseconds;
	LARGE_INTEGER Frequency;

	//thread pro part.
	int p = world->getAllPartNum();

	int blockSize = 128;
	int numThreads = (int)fmin(blockSize, p);
	int numBlocks = nearHighVal(p, numThreads);

	QueryPerformanceFrequency(&Frequency);
	QueryPerformanceCounter(&StartingTime);

	updateGridC <<< numBlocks, numThreads >>>(gridCounters, gridCells, pPos, pGridIndex, p);	// voxelSL, gridSL, , gridMinPosVec
	hipDeviceSynchronize();

	QueryPerformanceCounter(&EndingTime);
	ElapsedMicroseconds.QuadPart = EndingTime.QuadPart - StartingTime.QuadPart;
	//cout << ElapsedMicroseconds.QuadPart << endl;
	ElapsedMicroseconds.QuadPart *= 1000000;
	ElapsedMicroseconds.QuadPart /= Frequency.QuadPart;
	//cout << ElapsedMicroseconds.QuadPart << endl;
	demo->file2 << ElapsedMicroseconds.QuadPart * 0.000001 << endl;
}

//<<<<<<<<<< rigidbody kernels >>>>>>>>>>
void updateMom(float* rbMass, glm::vec3* rbForce, glm::vec3* rbPos, glm::vec3* rbLinMom, glm::vec3* rbAngMom, glm::vec3* pPos, glm::vec3* pForce){	//, float duration, float termVeloc

	LARGE_INTEGER StartingTime, EndingTime, ElapsedMicroseconds;
	LARGE_INTEGER Frequency;

	//thread pro body
	int b = world->getAllBodyNum();
	int blockSize = 128;
	int numThreads = (int)fmin(blockSize, b);
	int numBlocks = nearHighVal(b, numThreads);

	QueryPerformanceFrequency(&Frequency);
	QueryPerformanceCounter(&StartingTime);

	updateMomC<<<numBlocks, numThreads>>>(rbMass, rbForce, rbPos, rbLinMom, rbAngMom, pPos, pForce, b);	//duration, termVeloc,
	hipDeviceSynchronize();

	QueryPerformanceCounter(&EndingTime);
	ElapsedMicroseconds.QuadPart = EndingTime.QuadPart - StartingTime.QuadPart;
	//cout << ElapsedMicroseconds.QuadPart << endl;
	ElapsedMicroseconds.QuadPart *= 1000000;
	ElapsedMicroseconds.QuadPart /= Frequency.QuadPart;
	//cout << ElapsedMicroseconds.QuadPart << endl;
	demo->file3 << ElapsedMicroseconds.QuadPart * 0.000001 << endl;
}

void iterate(float* rbMass, glm::vec3* rbPos, glm::vec3* rbVeloc, glm::vec3* rbLinMom, glm::quat* rbRotQuat, glm::mat3* rbRotMat, glm::vec3* rbAngVeloc, glm::vec3* rbAngMom, glm::vec3* initIITDiago, glm::mat3* inverInertTens){	//float duration, float pRadius

	LARGE_INTEGER StartingTime, EndingTime, ElapsedMicroseconds;
	LARGE_INTEGER Frequency;

	//thread pro body
	int b = world->getAllBodyNum();
	int blockSize = 128;
	int numThreads = (int)fmin(blockSize, b);
	int numBlocks = nearHighVal(b, numThreads);

	QueryPerformanceFrequency(&Frequency);
	QueryPerformanceCounter(&StartingTime);

	iterateC<<<numBlocks, numThreads>>>(rbMass, rbPos, rbVeloc, rbLinMom, rbRotQuat, rbRotMat, rbAngVeloc, rbAngMom, initIITDiago, inverInertTens,b);	//, duration, pRadius
	hipDeviceSynchronize();

	QueryPerformanceCounter(&EndingTime);
	ElapsedMicroseconds.QuadPart = EndingTime.QuadPart - StartingTime.QuadPart;
	//cout << ElapsedMicroseconds.QuadPart << endl;
	ElapsedMicroseconds.QuadPart *= 1000000;
	ElapsedMicroseconds.QuadPart /= Frequency.QuadPart;
	//cout << ElapsedMicroseconds.QuadPart << endl;
	demo->file4 << ElapsedMicroseconds.QuadPart * 0.000001 << endl;
}

//<<<<<<<<<< particles kernels >>>>>>>>>>
void calcCollForces(float* pMass, glm::vec3* pPos, glm::vec3* pVeloc, glm::vec3* pForce, glm::ivec3* pGridIndex, int* gridCounters, glm::ivec4* gridCells){	//, int gridSL , float pRadius, float worldS, float springC, float dampC

	LARGE_INTEGER StartingTime, EndingTime, ElapsedMicroseconds;
	LARGE_INTEGER Frequency;

	//thread pro part.
	int p = world->getAllPartNum();
		
	int blockSize = 128;
	int numThreads = (int)fmin(blockSize, p);
	int numBlocks = nearHighVal(p, numThreads);
	
	QueryPerformanceFrequency(&Frequency);
	QueryPerformanceCounter(&StartingTime);

	calcCollForcesC <<< numBlocks, numThreads >>>(pMass, pPos, pVeloc, pForce, pGridIndex, gridCounters, gridCells, p);	// gridSL, , pRadius, worldS, springC, dampC
	hipDeviceSynchronize();

	QueryPerformanceCounter(&EndingTime);
	ElapsedMicroseconds.QuadPart = EndingTime.QuadPart - StartingTime.QuadPart;
	//cout << ElapsedMicroseconds.QuadPart << endl;
	ElapsedMicroseconds.QuadPart *= 1000000;
	ElapsedMicroseconds.QuadPart /= Frequency.QuadPart;
	//cout << ElapsedMicroseconds.QuadPart << endl;
	demo->file5 << ElapsedMicroseconds.QuadPart * 0.000001 << endl;
}

void updatePart(glm::vec3* rbPos, glm::vec3* rbVeloc, glm::mat3* rbRotMat, glm::vec3* rbAngVeloc, glm::vec3* pPos, glm::vec3* pVeloc){	//, float pRadius

	LARGE_INTEGER StartingTime, EndingTime, ElapsedMicroseconds;
	LARGE_INTEGER Frequency;

	//threads pro part.
	int p = world->getAllPartNum();
	int blockSize = 128;
	int numThreads = (int)fmin(blockSize, p);
	int numBlocks = nearHighVal(p, numThreads);

	QueryPerformanceFrequency(&Frequency);
	QueryPerformanceCounter(&StartingTime);

	updatePartC <<< numBlocks, numThreads >>>(rbPos, rbVeloc, rbRotMat, rbAngVeloc, pPos, pVeloc, p);	//pRadius,
	hipDeviceSynchronize();

	QueryPerformanceCounter(&EndingTime);
	ElapsedMicroseconds.QuadPart = EndingTime.QuadPart - StartingTime.QuadPart;
	//cout << ElapsedMicroseconds.QuadPart << endl;
	ElapsedMicroseconds.QuadPart *= 1000000;
	ElapsedMicroseconds.QuadPart /= Frequency.QuadPart;
	//cout << ElapsedMicroseconds.QuadPart << endl;
	demo->file6 << ElapsedMicroseconds.QuadPart * 0.000001 << endl;
}
