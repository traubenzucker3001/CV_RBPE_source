#include "hip/hip_runtime.h"
//neuer cuda file zum l�sen der include/compiler fehler

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "UniformGrid.h"
#include "World.h"

#include "kernels_impl.cuh"

//link fix try 3
extern int allPartNum;
extern int allBodyNum;

int nearHighVal(int a, int b){
	return (a % b != 0) ? (a / b + 1) : (a / b);
}	//nicht in jedem cu eine funktion, irgwo f�r alle erreichbar machen

//extern "C"{
	//<<<<<<<<<< uniformgrid kernels >>>>>>>>>>
	void resetGrid(int* countGrid, glm::vec4* indexGrid){

		//blocks und threads berechn.
		//int b = World::getInstance()->getAllBodyNum();	//wird bodies oder particle ben�tigt, oder gitter abh�ngiges
		//thread pro gitterzelle	//wie komm ich an diese zahl?!
		int g = UniformGrid::getInstance()->getGridSize();
		int blockSize = 64;
		int numThreads = (int)fmin(blockSize, g);
		int numBlocks = nearHighVal(g, numThreads);
		//geht doch bestimmt auch noch "besser"!!?

		resetGridC <<< numBlocks, numThreads >>>(countGrid, indexGrid);
	}

	//updateGRid
	void updateGrid(int* countGrid, glm::vec4* indexGrid, glm::vec3* pPos, glm::vec3 gridMinPosVec, float voxelSL, int gridSL, glm::vec3* pGridIndex){

		//blocks und threads berechn.
		//int b = World::getInstance()->getAllBodyNum();	//wird bodies oder particle ben�tigt, oder gitter abh�ngiges
		//thread pro part.
		//int p = World::getInstance()->getAllPartNum();
		int p = allPartNum;
		int blockSize = 64;
		int numThreads = (int)fmin(blockSize, p);
		int numBlocks = nearHighVal(p, numThreads);
		//geht doch bestimmt auch noch "besser"!!?

		updateGridC <<< numBlocks, numThreads >>>(countGrid, indexGrid, pPos, gridMinPosVec, voxelSL, gridSL, pGridIndex);
	}

	//<<<<<<<<<< rigidbody kernels >>>>>>>>>>
	//update momenta
	void updateMom(float* rbMass, glm::vec3* rbForce, glm::vec3* rbPos, glm::vec3* rbLinMom, glm::vec3* rbAngMom, glm::vec3* pPos, glm::vec3* pForce, float duration, float termVeloc){

		//todo: blocks und threads berechn.
		//thread pro body
		//int b = World::getInstance()->getAllBodyNum();
		int b = allBodyNum;
		int blockSize = 64;
		int numThreads = (int)fmin(blockSize, b);
		int numBlocks = nearHighVal(b, numThreads);
		//geht doch bestimmt auch noch "besser"!!?

		updateMomC <<< numBlocks, numThreads >>>(rbMass, rbForce, rbPos, rbLinMom, rbAngMom, pPos, pForce, duration, termVeloc);
	}

	//perform step
	void iterate(float* rbMass, glm::vec3* rbPos, glm::vec3* rbVeloc, glm::vec3* rbLinMom, glm::quat* rbRotQuat, glm::mat3* rbRotMat, glm::vec3* rbAngVeloc, glm::vec3* rbAngMom, glm::vec3* initIITDiago, glm::mat3* inverInertTens, float duration, float pRadius){

		//todo: blocks und threads berechn.
		//thread pro body
		//int b = World::getInstance()->getAllBodyNum();
		int b = allBodyNum;
		int blockSize = 64;
		int numThreads = (int)fmin(blockSize, b);
		int numBlocks = nearHighVal(b, numThreads);
		//geht doch bestimmt auch noch "besser"!!?

		iterateC <<< numBlocks, numThreads >>>(rbMass, rbPos, rbVeloc, rbLinMom, rbRotQuat, rbRotMat, rbAngVeloc, rbAngMom, initIITDiago, inverInertTens, duration, pRadius);
	}

	//<<<<<<<<<< particles kernels >>>>>>>>>>
	//collision detection
	void calcCollForces(float* pMass, glm::vec3* pPos, glm::vec3* pVeloc, glm::vec3* pForce, float pRadius, float worldS, float springC, float dampC, glm::vec3* pGridIndex, int* countGrid, glm::vec4* indexGrid, int gridSL){

		//blocks und threads berechn.
		//thread pro part.
		//int p = World::getInstance()->getAllPartNum();
		int p = allPartNum;
		int blockSize = 64;
		int numThreads = (int)fmin(blockSize, p);
		int numBlocks = nearHighVal(p, numThreads);
		//geht doch bestimmt auch noch "besser"!!?

		calcCollForcesC <<< numBlocks, numThreads >>>(pMass, pPos, pVeloc, pForce, pRadius, worldS, springC, dampC, pGridIndex, countGrid, indexGrid, gridSL);
	}

	void updatePart(glm::vec3* rbPos, glm::vec3* rbVeloc, glm::mat3* rbRotMat, glm::vec3* rbAngVeloc, glm::vec3* pPos, glm::vec3* pVeloc, float pRadius){

		//blocks und threads berechn.
		//threads pro part.
		//int p = World::getInstance()->getAllPartNum();
		int p = allPartNum;
		int blockSize = 64;
		int numThreads = (int)fmin(blockSize, p);
		int numBlocks = nearHighVal(p, numThreads);
		//geht doch bestimmt auch noch "besser"!!?

		updatePartC <<< numBlocks, numThreads >>>(rbPos, rbVeloc, rbRotMat, rbAngVeloc, pPos, pVeloc, pRadius);
	}

//}