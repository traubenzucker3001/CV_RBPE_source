#include "hip/hip_runtime.h"
//neuer cuda file zum l�sen der include/compiler fehler

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "UniformGrid.h"
#include "World.h"

#include "kernels_impl.cuh"

//link fix try 4
extern World* world;

int nearHighVal(int a, int b){
	return (a % b != 0) ? (a / b + 1) : (a / b);
}	//nicht in jedem cu eine funktion, irgwo f�r alle erreichbar machen

//extern "C"{
	//<<<<<<<<<< uniformgrid kernels >>>>>>>>>>
	void resetGrid(int* countGrid, glm::vec4* indexGrid){
		cout << "-test reset grid 1-" << endl; //zum debuggen
		//blocks und threads berechn.
		//int b = World::getInstance()->getAllBodyNum();	//wird bodies oder particle ben�tigt, oder gitter abh�ngiges
		//thread pro gitterzelle	//wie komm ich an diese zahl?!
		int g = UniformGrid::getInstance()->getGridSize();	//gridsize=0, why?!
		cout << "-test reset grid 1.3- g: " << g << endl; //zum debuggen
		int blockSize = 64;
		cout << "-test reset grid 1.5-" << endl; //zum debuggen
		int numThreads = (int)fmin(blockSize, g);
		cout << "-test reset grid 1.7- threads: " << numThreads << endl; //zum debuggen
		int numBlocks = nearHighVal(g, numThreads);		//<--- fehler dort
		//geht doch bestimmt auch noch "besser"!!?
		cout << "-test reset grid 2-" << endl; //zum debuggen
		resetGridC <<< numBlocks, numThreads >>>(countGrid, indexGrid);
		cout << "-test reset grid 3-" << endl; //zum debuggen
	}

	//updateGRid
	void updateGrid(int* countGrid, glm::vec4* indexGrid, glm::vec3* pPos, glm::vec3 gridMinPosVec, float voxelSL, int gridSL, glm::vec3* pGridIndex){

		//blocks und threads berechn.
		//int b = World::getInstance()->getAllBodyNum();	//wird bodies oder particle ben�tigt, oder gitter abh�ngiges
		//thread pro part.
		int p = world->getAllPartNum();
		int blockSize = 64;
		int numThreads = (int)fmin(blockSize, p);
		int numBlocks = nearHighVal(p, numThreads);
		//geht doch bestimmt auch noch "besser"!!?

		updateGridC <<< numBlocks, numThreads >>>(countGrid, indexGrid, pPos, gridMinPosVec, voxelSL, gridSL, pGridIndex);
	}

	//<<<<<<<<<< rigidbody kernels >>>>>>>>>>
	//update momenta
	void updateMom(float* rbMass, glm::vec3* rbForce, glm::vec3* rbPos, glm::vec3* rbLinMom, glm::vec3* rbAngMom, glm::vec3* pPos, glm::vec3* pForce, float duration, float termVeloc){

		//todo: blocks und threads berechn.
		//thread pro body
		int b = world->getAllBodyNum();
		int blockSize = 64;
		int numThreads = (int)fmin(blockSize, b);
		int numBlocks = nearHighVal(b, numThreads);
		//geht doch bestimmt auch noch "besser"!!?

		updateMomC <<< numBlocks, numThreads >>>(rbMass, rbForce, rbPos, rbLinMom, rbAngMom, pPos, pForce, duration, termVeloc);
	}

	//perform step
	void iterate(float* rbMass, glm::vec3* rbPos, glm::vec3* rbVeloc, glm::vec3* rbLinMom, glm::quat* rbRotQuat, glm::mat3* rbRotMat, glm::vec3* rbAngVeloc, glm::vec3* rbAngMom, glm::vec3* initIITDiago, glm::mat3* inverInertTens, float duration, float pRadius){

		//todo: blocks und threads berechn.
		//thread pro body
		int b = world->getAllBodyNum();
		int blockSize = 64;
		int numThreads = (int)fmin(blockSize, b);
		int numBlocks = nearHighVal(b, numThreads);
		//geht doch bestimmt auch noch "besser"!!?

		iterateC <<< numBlocks, numThreads >>>(rbMass, rbPos, rbVeloc, rbLinMom, rbRotQuat, rbRotMat, rbAngVeloc, rbAngMom, initIITDiago, inverInertTens, duration, pRadius);
	}

	//<<<<<<<<<< particles kernels >>>>>>>>>>
	//collision detection
	void calcCollForces(float* pMass, glm::vec3* pPos, glm::vec3* pVeloc, glm::vec3* pForce, float pRadius, float worldS, float springC, float dampC, glm::vec3* pGridIndex, int* countGrid, glm::vec4* indexGrid, int gridSL){

		//blocks und threads berechn.
		//thread pro part.
		int p = world->getAllPartNum();
		int blockSize = 64;
		int numThreads = (int)fmin(blockSize, p);
		int numBlocks = nearHighVal(p, numThreads);
		//geht doch bestimmt auch noch "besser"!!?

		calcCollForcesC <<< numBlocks, numThreads >>>(pMass, pPos, pVeloc, pForce, pRadius, worldS, springC, dampC, pGridIndex, countGrid, indexGrid, gridSL);
	}

	void updatePart(glm::vec3* rbPos, glm::vec3* rbVeloc, glm::mat3* rbRotMat, glm::vec3* rbAngVeloc, glm::vec3* pPos, glm::vec3* pVeloc, float pRadius){

		//blocks und threads berechn.
		//threads pro part.
		int p = world->getAllPartNum();
		int blockSize = 64;
		int numThreads = (int)fmin(blockSize, p);
		int numBlocks = nearHighVal(p, numThreads);
		//geht doch bestimmt auch noch "besser"!!?

		updatePartC <<< numBlocks, numThreads >>>(rbPos, rbVeloc, rbRotMat, rbAngVeloc, pPos, pVeloc, pRadius);
	}

//}