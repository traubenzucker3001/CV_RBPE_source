#include "hip/hip_runtime.h"
//neuer cuda file zum l�sen der include/compiler fehler

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define GLM_FORCE_CUDA
#include <glm\glm.hpp>

//#include "UniformGrid.h"
#include "World.h"

#include "kernels.cuh"
#include "kernels_impl.cuh"

//link fix try 4
extern World* world;

hipError_t err = hipSuccess;	//zum debuggen

int nearHighVal(int a, int b){
	return (a % b != 0) ? (a / b + 1) : (a / b);
}

void fillDeviceSymbols(float voxelS, int gridSL, float worldS, float springC, float dampC, float pRadius, float duration, float termVeloc, glm::vec3 gridMinPosVector){
	//siehe cuda programming guide. sollte eig ohne "" gehen
	err = hipMemcpyToSymbol((const void*)&d_voxelS, &voxelS, sizeof(float));
	if (err != hipSuccess){																				//zum debuggen
		fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n", hipGetErrorString(err));	//zum debuggen
		//exit(EXIT_FAILURE);
	}																										//zum debuggen
	err = hipSuccess;																						//zum debuggen
	//hipMemcpyToSymbol(HIP_SYMBOL("d_gridS"), &h_gridS, sizeof(int));
	err = hipMemcpyToSymbol((const void*)&d_gridSL, &gridSL, sizeof(int));
	if (err != hipSuccess){																				//zum debuggen
		fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n", hipGetErrorString(err));	//zum debuggen
		//exit(EXIT_FAILURE);
	}																										//zum debuggen
	err = hipSuccess;																						//zum debuggen
	err = hipMemcpyToSymbol((const void*)&d_worldS, &worldS, sizeof(float));
	if (err != hipSuccess){																				//zum debuggen
		fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n", hipGetErrorString(err));	//zum debuggen
		//exit(EXIT_FAILURE);
	}																										//zum debuggen
	err = hipSuccess;																						//zum debuggen
	err = hipMemcpyToSymbol((const void*)&d_springC, &springC, sizeof(float));
	if (err != hipSuccess){																				//zum debuggen
		fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n", hipGetErrorString(err));	//zum debuggen
		//exit(EXIT_FAILURE);
	}																										//zum debuggen
	err = hipSuccess;																						//zum debuggen
	err = hipMemcpyToSymbol((const void*)&d_dampC, &dampC, sizeof(float));
	if (err != hipSuccess){																				//zum debuggen
		fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n", hipGetErrorString(err));	//zum debuggen
		//exit(EXIT_FAILURE);
	}																										//zum debuggen
	err = hipSuccess;																						//zum debuggen
	err = hipMemcpyToSymbol((const void*)&d_pRadius, &pRadius, sizeof(float));
	if (err != hipSuccess){																				//zum debuggen
		fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n", hipGetErrorString(err));	//zum debuggen
		//exit(EXIT_FAILURE);
	}																										//zum debuggen
	err = hipSuccess;																						//zum debuggen
	err = hipMemcpyToSymbol((const void*)&d_duration, &duration, sizeof(float));
	if (err != hipSuccess){																				//zum debuggen
		fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n", hipGetErrorString(err));	//zum debuggen
		//exit(EXIT_FAILURE);
	}																										//zum debuggen
	err = hipSuccess;																						//zum debuggen
	err = hipMemcpyToSymbol((const void*)&d_termVeloc, &termVeloc, sizeof(float));
	if (err != hipSuccess){																				//zum debuggen
		fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n", hipGetErrorString(err));	//zum debuggen
		//exit(EXIT_FAILURE);
	}																										//zum debuggen
	err = hipSuccess;																						//zum debuggen

	//hipMemcpyToSymbol(HIP_SYMBOL("d_gridMinPosVector"), &h_gridMinPosVector, sizeof(glm::vec3));
	err = hipMemcpyToSymbol((const void*)&d_gridMinPosVecX, &gridMinPosVector.x, sizeof(float));
	if (err != hipSuccess){																				//zum debuggen
		fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n", hipGetErrorString(err));	//zum debuggen
		//exit(EXIT_FAILURE);
	}																										//zum debuggen
	err = hipSuccess;																						//zum debuggen
	err = hipMemcpyToSymbol((const void*)&d_gridMinPosVecY, &gridMinPosVector.y, sizeof(float));
	if (err != hipSuccess){																				//zum debuggen
		fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n", hipGetErrorString(err));	//zum debuggen
		//exit(EXIT_FAILURE);
	}																										//zum debuggen
	err = hipSuccess;																						//zum debuggen
	err = hipMemcpyToSymbol((const void*)&d_gridMinPosVecZ, &gridMinPosVector.z, sizeof(float));
	if (err != hipSuccess){																				//zum debuggen
		fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n", hipGetErrorString(err));	//zum debuggen
		//exit(EXIT_FAILURE);
	}																										//zum debuggen
	err = hipSuccess;																						//zum debuggen
}

//extern "C"{
	//<<<<<<<<<< uniformgrid kernels >>>>>>>>>>
		void resetGrid(int* gridCounters, glm::ivec4* gridCells, int g){

		//blocks und threads berechn.
		//thread pro gitterzelle	//wie komm ich an diese zahl?!
		//int g = UniformGrid::getInstance()->getGridSize();	//gridsize=0, why?!
		cout << "gridsize: " << g << endl;	//zum debuggen
		int blockSize = 64;	//64, 256, 512, 1024
		//int numThreads = (int)fmin(blockSize, g);
		int numThreads = blockSize;
		int numBlocks = nearHighVal(g, numThreads);	
		//int numBlocks = ((g + 1024 -1) / 1024);
		cout << "threads: " << numThreads << endl;	//zum debuggen
		cout << "blocks: " << numBlocks << endl;	//zum debuggen
		//geht doch bestimmt auch noch "besser"!!?
		//int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;	//aus vectorAdd

		resetGridC <<< numBlocks, numThreads >>>(gridCounters, gridCells, g);
		hipDeviceSynchronize();
	}

	//updateGRid
	void updateGrid(int* gridCounters, glm::ivec4* gridCells, glm::vec3* pPos, glm::ivec3* pGridIndex){	//, float voxelSL, int gridSL , glm::vec3 gridMinPosVec

		//blocks und threads berechn.
		//int b = World::getInstance()->getAllBodyNum();	//wird bodies oder particle ben�tigt, oder gitter abh�ngiges
		//thread pro part.
		int p = world->getAllPartNum();
		//cout << "gridsize: " << p << endl;	//zum debuggen
		int blockSize = 64;
		int numThreads = (int)fmin(blockSize, p);
		int numBlocks = nearHighVal(p, numThreads);
		//cout << "threads: " << numThreads << endl;	//zum debuggen
		//cout << "blocks: " << numBlocks << endl;	//zum debuggen
		//geht doch bestimmt auch noch "besser"!!?

		updateGridC <<< numBlocks, numThreads >>>(gridCounters, gridCells, pPos, pGridIndex, p);	// voxelSL, gridSL, , gridMinPosVec
		hipDeviceSynchronize();
	}

	//<<<<<<<<<< rigidbody kernels >>>>>>>>>>
	//update momenta
	void updateMom(float* rbMass, glm::vec3* rbForce, glm::vec3* rbPos, glm::vec3* rbLinMom, glm::vec3* rbAngMom, glm::vec3* pPos, glm::vec3* pForce){	//, float duration, float termVeloc

		//todo: blocks und threads berechn.
		//thread pro body
		int b = world->getAllBodyNum();
		int blockSize = 64;
		int numThreads = (int)fmin(blockSize, b);
		int numBlocks = nearHighVal(b, numThreads);
		//geht doch bestimmt auch noch "besser"!!?

		updateMomC<<<numBlocks, numThreads>>>(rbMass, rbForce, rbPos, rbLinMom, rbAngMom, pPos, pForce, b);	//duration, termVeloc,
		hipDeviceSynchronize();
	}

	//perform step
	void iterate(float* rbMass, glm::vec3* rbPos, glm::vec3* rbVeloc, glm::vec3* rbLinMom, glm::quat* rbRotQuat, glm::mat3* rbRotMat, glm::vec3* rbAngVeloc, glm::vec3* rbAngMom, glm::vec3* initIITDiago, glm::mat3* inverInertTens){	//float duration, float pRadius

		//todo: blocks und threads berechn.
		//thread pro body
		int b = world->getAllBodyNum();
		int blockSize = 64;
		int numThreads = (int)fmin(blockSize, b);
		int numBlocks = nearHighVal(b, numThreads);
		//geht doch bestimmt auch noch "besser"!!?

		iterateC<<<numBlocks, numThreads>>>(rbMass, rbPos, rbVeloc, rbLinMom, rbRotQuat, rbRotMat, rbAngVeloc, rbAngMom, initIITDiago, inverInertTens,b);	//, duration, pRadius
		hipDeviceSynchronize();
	}

	//<<<<<<<<<< particles kernels >>>>>>>>>>
	//collision detection
	void calcCollForces(float* pMass, glm::vec3* pPos, glm::vec3* pVeloc, glm::vec3* pForce, glm::ivec3* pGridIndex, int* gridCounters, glm::ivec4* gridCells){	//, int gridSL , float pRadius, float worldS, float springC, float dampC

		//blocks und threads berechn.
		//thread pro part.
		int p = world->getAllPartNum();
		cout << "partnum: " << p << endl;	//zum debuggen
		int blockSize = 512;
		int numThreads = (int)fmin(blockSize, p);
		int numBlocks = nearHighVal(p, numThreads);
		//geht doch bestimmt auch noch "besser"!!?
		cout << "threads: " << numThreads << endl;	//zum debuggen
		cout << "blocks: " << numBlocks << endl;	//zum debuggen

		calcCollForcesC <<< numBlocks, numThreads >>>(pMass, pPos, pVeloc, pForce, pGridIndex, gridCounters, gridCells, p);	// gridSL, , pRadius, worldS, springC, dampC
		hipDeviceSynchronize();
	}

	void updatePart(glm::vec3* rbPos, glm::vec3* rbVeloc, glm::mat3* rbRotMat, glm::vec3* rbAngVeloc, glm::vec3* pPos, glm::vec3* pVeloc){	//, float pRadius

		//blocks und threads berechn.
		//threads pro part.
		int p = world->getAllPartNum();
		int blockSize = 64;
		int numThreads = (int)fmin(blockSize, p);
		int numBlocks = nearHighVal(p, numThreads);
		//geht doch bestimmt auch noch "besser"!!?

		updatePartC <<< numBlocks, numThreads >>>(rbPos, rbVeloc, rbRotMat, rbAngVeloc, pPos, pVeloc, p);	//pRadius,
		hipDeviceSynchronize();
	}

//}